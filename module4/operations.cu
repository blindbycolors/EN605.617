#include "hip/hip_runtime.h"
//
// Created by nou on 2/21/21.
//

#include <cstdio>
#include "operations.cuh"

#define ACTIVATE_OP_KERNEL(opFunc, numBlocks, blockSize, d_a, d_b, d_out)      \
    hipEvent_t kernelStart, kernelStop;                                       \
    auto delta = 0.0F;                                                         \
    checkCuda(hipEventCreate(&kernelStart));                                  \
    checkCuda(hipEventCreate(&kernelStop));                                   \
    checkCuda(hipEventRecord(kernelStart,0));                                 \
    opFunc<<<numBlocks, blockSize>>>(d_a, d_b, d_out);                         \
    checkCuda(hipEventRecord(kernelStop,0));                                  \
    checkCuda(hipEventSynchronize(kernelStop));                               \
    checkCuda(hipEventElapsedTime(&delta, kernelStart, kernelStop));          \
    checkCuda(hipEventDestroy(kernelStart));                                  \
    checkCuda(hipEventDestroy(kernelStop));

#define ACTIVATE_CAESER_KERNEL(opFunction, numBlocks, blockSize, totalThreads, \
                               d_a, d_out, offset)                             \
    hipEvent_t kernelStart, kernelStop;                                       \
    auto delta = 0.0F;                                                         \
    checkCuda(hipEventCreate(&kernelStart));                                  \
    checkCuda(hipEventCreate(&kernelStop));                                   \
    checkCuda(hipEventRecord(kernelStart, 0));                                \
    opFunction<<<numBlocks, blockSize>>>(d_a, totalThreads, offset, d_out);    \
    checkCuda(hipEventRecord(kernelStop, 0));                                 \
    checkCuda(hipEventSynchronize(kernelStop));                               \
    checkCuda(hipEventElapsedTime(&delta, kernelStart, kernelStop));          \
    checkCuda(hipEventDestroy(kernelStart));                                  \
    checkCuda(hipEventDestroy(kernelStop));

#define DEVICE_ALLOCATE(a, type, totalThreads)                                 \
    type *a;                                                                   \
    checkCuda(hipMalloc((void**) &a, sizeof(type) * totalThreads));

__global__ void kernel_addition(const int *a, const int *b, float *out)
{
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    out[tid] = a[tid] + b[tid];
}

__global__ void kernel_subtraction(const int *a, const int *b, float *out)
{
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    out[tid] = a[tid] - b[tid];
}

__global__ void kernel_multiplication(const int *a, const int *b, float *out)
{
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    out[tid] = a[tid] * b[tid];
}

__global__ void kernel_modulus(const int *a, const int *b, float *out)
{
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    out[tid] = (float) (a[tid] % b[tid]);
}

__global__ void kernel_caeser_encrypt(const char *value, const int n,
                                      const int offset, char *d_out)
{
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n)
    {
        auto encryptedVal = value[tid] + offset % 128;
        d_out[tid] = encryptedVal;
    }
}

__global__ void kernel_caeser_decrypt(const char *value, const int n,
                                      const int offset, char *d_out)
{
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n)
    {
        auto decryptedVal = value[tid] - offset % 128 ;
        d_out[tid] = decryptedVal;
    }
}


float hostAdd(const int numBlocks, const int blockSize,
              const int totalThreads, int *h_a, int *h_b)
{
    // Device Array
    DEVICE_ALLOCATE(d_out, float, totalThreads);
    DEVICE_ALLOCATE(d_a, int, totalThreads);
    DEVICE_ALLOCATE(d_b, int, totalThreads);

    // Copy from host to device
    checkCuda(hipMemcpy(d_a, h_a, sizeof(int) * totalThreads,
                         hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_b, h_b, sizeof(int) * totalThreads,
                         hipMemcpyHostToDevice));

    // Run the kernel function
    ACTIVATE_OP_KERNEL(kernel_addition, numBlocks, blockSize, d_a, d_b,
                       d_out);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    return delta;
}

float hostSubtract(const int numBlocks, const int blockSize,
                   const int totalThreads, int *h_a, int *h_b)
{
    // Device Array
    DEVICE_ALLOCATE(d_out, float, totalThreads);
    DEVICE_ALLOCATE(d_a, int, totalThreads);
    DEVICE_ALLOCATE(d_b, int, totalThreads);

    // Copy from host to device
    checkCuda(hipMemcpy(d_a, h_a, sizeof(int) * totalThreads,
                         hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_b, h_b, sizeof(int) * totalThreads,
                         hipMemcpyHostToDevice));

    // Run the kernel function
    ACTIVATE_OP_KERNEL(kernel_subtraction, numBlocks, blockSize, d_a, d_b,
                       d_out);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    return delta;
}


float hostMultiply(const int numBlocks, const int blockSize,
                   const int totalThreads, int *h_a, int *h_b)
{
    // Device Array
    DEVICE_ALLOCATE(d_out, float, totalThreads);
    DEVICE_ALLOCATE(d_a, int, totalThreads);
    DEVICE_ALLOCATE(d_b, int, totalThreads);

    // Copy from host to device
    checkCuda(hipMemcpy(d_a, h_a, sizeof(int) * totalThreads,
                         hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_b, h_b, sizeof(int) * totalThreads,
                         hipMemcpyHostToDevice));

    // Run the kernel function
    ACTIVATE_OP_KERNEL(kernel_multiplication, numBlocks, blockSize, d_a, d_b,
                       d_out);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    return delta;
}

float hostMod(const int numBlocks, const int blockSize,
                     const int totalThreads, int *h_a, int *h_b)
{
    // Device Array
    DEVICE_ALLOCATE(d_out, float, totalThreads);
    DEVICE_ALLOCATE(d_a, int, totalThreads);
    DEVICE_ALLOCATE(d_b, int, totalThreads);

    // Copy from host to device
    checkCuda(hipMemcpy(d_a, h_a, sizeof(int) * totalThreads,
                         hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_b, h_b, sizeof(int) * totalThreads,
                         hipMemcpyHostToDevice));

    // Run the kernel function
    ACTIVATE_OP_KERNEL(kernel_modulus, numBlocks, blockSize, d_a, d_b, d_out);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    return delta;
}

std::string
hostEncrypt(const int numBlocks, const int blockSize,
            const char *h_aInput, int strLength, int offset)
{
    DEVICE_ALLOCATE(d_a, char, strLength);
    DEVICE_ALLOCATE(d_out, char, strLength);

    auto h_outEncrypt = (char *) malloc(sizeof(char) * strLength);
    checkCuda(hipMemcpy(d_a, h_aInput, sizeof(char) * strLength,
                         hipMemcpyHostToDevice));

    ACTIVATE_CAESER_KERNEL(kernel_caeser_encrypt, numBlocks, blockSize,
                           strLength, d_a, d_out, offset);

    printf("Encryption Duration: %f ms\n", delta);

    checkCuda(hipMemcpy(h_outEncrypt, d_out, sizeof(char) * strLength,
                         hipMemcpyDeviceToHost));
    std::string retString(h_outEncrypt);

    hipFree(d_a);
    hipFree(d_out);
    free(h_outEncrypt);

    return retString;
}

std::string
hostDecrypt(const int numBlocks, const int blockSize,
            const char *h_aInput, int strLength, int offset)
{
    DEVICE_ALLOCATE(d_a, char, strLength);
    DEVICE_ALLOCATE(d_out, char, strLength);

    auto h_outEncrypt = (char *) malloc(sizeof(char) * strLength);
    checkCuda(hipMemcpy(d_a, h_aInput, sizeof(char) * strLength,
                         hipMemcpyHostToDevice));
    ACTIVATE_CAESER_KERNEL(kernel_caeser_decrypt, numBlocks, blockSize,
                           strLength, d_a, d_out, offset);

    printf("Decryption Duration: %f ms\n", delta);

    checkCuda(hipMemcpy(h_outEncrypt, d_out, sizeof(char) * strLength,
                         hipMemcpyDeviceToHost));

    std::string retString(h_outEncrypt);

    hipFree(d_a);
    hipFree(d_out);
    free(h_outEncrypt);

    return retString;
}
